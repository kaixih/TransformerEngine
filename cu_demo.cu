#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include "common/util/vectorized_pointwise.h"
#include "common/gemm/cublaslt_gemm.h"

#define checkCUDA(expression)                              \
{                                                          \
  hipError_t status = (expression);                       \
  if (status != hipSuccess) {                             \
    std::cerr << "Error on line " << __LINE__ << ": "      \
              << hipGetErrorString(status) << std::endl;  \
    std::exit(EXIT_FAILURE);                               \
  }                                                        \
}

using byte = uint8_t;
using int32 = int32_t;
using fp32 = float;
using fp16 = half;
using bf16 = hip_bfloat16;
using fp8e4m3 = __hip_fp8_e4m3_fnuz;
using fp8e5m2 = __hip_fp8_e5m2_fnuz;

namespace detail {

struct Empty {};

__device__ inline fp32 identity(fp32 value, const Empty&) {
  return value;
}

}  // namespace detail

extern "C" {

void cast_to_fp8(const void* input_ptr, const void* scale_ptr, void *amax_ptr,
                 void *scale_inv_ptr, void* output_ptr, size_t dim0,
                 size_t dim1, int dtype) {

  const size_t N = dim0 * dim1;
  using IType = float;
  using OType = fp8e4m3;
  IType* input = nullptr;
  OType* output = nullptr;
  fp32* scale = nullptr;
  fp32* scale_inv = nullptr;
  fp32* amax = nullptr;
  checkCUDA(hipMalloc((void**)&input, N * sizeof(IType)));
  checkCUDA(hipMalloc((void**)&output, N * sizeof(OType)));
  checkCUDA(hipMalloc((void**)&scale, 1 * sizeof(fp32)));
  checkCUDA(hipMalloc((void**)&scale_inv, 1 * sizeof(fp32)));
  checkCUDA(hipMalloc((void**)&amax, 1 * sizeof(fp32)));
  checkCUDA(hipMemcpy(input, input_ptr, N * sizeof(IType),
                       hipMemcpyHostToDevice));
  checkCUDA(hipMemcpy(scale, scale_ptr, 1 * sizeof(fp32),
                       hipMemcpyHostToDevice));


  constexpr int nvec = 32 / sizeof(IType);
  hipStream_t stream = 0;
  if (dtype == 0) {
    transformer_engine::VectorizedUnaryKernelLauncher<
        nvec, detail::Empty, detail::identity>(input, output, scale, scale_inv,
                                               amax, N, {}, stream);
  } else {

    fp8e5m2* output_cast = reinterpret_cast<fp8e5m2*>(output);
    transformer_engine::VectorizedUnaryKernelLauncher<
        nvec, detail::Empty, detail::identity>(
            input, output_cast, scale, scale_inv, amax, N, {}, stream);
  }

  checkCUDA(hipMemcpy(output_ptr, output, N * sizeof(OType),
                       hipMemcpyDeviceToHost));
  checkCUDA(hipMemcpy(scale_inv_ptr, scale_inv, 1 * sizeof(fp32),
                       hipMemcpyDeviceToHost));
  checkCUDA(hipMemcpy(amax_ptr, amax, 1 * sizeof(fp32),
                       hipMemcpyDeviceToHost));
}

void fp8_gemm(const void* A_ptr,
              const void* A_scale_inverse_ptr,
              int A_dtype,
              const void* B_ptr,
              const void* B_scale_inverse_ptr,
              int B_dtype,
              void* D_ptr,
              int A_dim0, int A_dim1,
              int B_dim0, int B_dim1,
              bool transa, bool transb, bool grad,
              bool accumulate, bool use_split_accumulate) {

  const int m = transa ? A_dim0 : A_dim1;
  const int k = transa ? A_dim1 : A_dim0;
  const int n = transb ? B_dim1 : B_dim0;

  const size_t A_N = m * k;
  const size_t B_N = k * n;
  const size_t D_N = m * n;
  using AType = fp8e4m3;
  using DType = float;
  AType* A = nullptr;
  AType* B = nullptr;
  fp32* D = nullptr;
  fp32* A_scale_inv = nullptr;
  fp32* B_scale_inv = nullptr;
  fp32* bias_ptr = nullptr;
  int workspaceSize = 33'554'432;
  void* workspace = nullptr;
  checkCUDA(hipMalloc((void**)&A, A_N * sizeof(AType)));
  checkCUDA(hipMalloc((void**)&B, B_N * sizeof(AType)));
  checkCUDA(hipMalloc((void**)&D, D_N * sizeof(DType)));
  checkCUDA(hipMalloc((void**)&A_scale_inv, 1 * sizeof(fp32)));
  checkCUDA(hipMalloc((void**)&B_scale_inv, 1 * sizeof(fp32)));
  // checkCUDA(hipMalloc((void**)&bias_ptr, n * sizeof(bf16)));
  checkCUDA(hipMalloc((void**)&workspace, workspaceSize));
  checkCUDA(hipMemcpy(A, A_ptr, A_N * sizeof(AType),
                       hipMemcpyHostToDevice));
  checkCUDA(hipMemcpy(B, B_ptr, B_N * sizeof(AType),
                       hipMemcpyHostToDevice));
  checkCUDA(hipMemcpy(A_scale_inv, A_scale_inverse_ptr, 1 * sizeof(fp32),
                       hipMemcpyHostToDevice));
  checkCUDA(hipMemcpy(B_scale_inv, B_scale_inverse_ptr, 1 * sizeof(fp32),
                       hipMemcpyHostToDevice));


  int lda, ldb, ldd;
  if (transa && !transb) {  // TN
    lda = k;
    ldb = k;
    ldd = m;
  } else if (!transa && !transb) {  // NN
    lda = m;
    ldb = k;
    ldd = m;
  } else if (!transa && transb) {  // NT
    lda = m;
    ldb = n;
    ldd = m;
  } else {  // TT
    printf("TT layout not allowed.\n");
    exit(0);
  }

  auto A_type = HIP_R_8F_E4M3;
  auto B_type = HIP_R_8F_E4M3;
  if (A_dtype != 0 ) A_type = HIP_R_8F_E5M2;
  if (B_dtype != 0 ) B_type = HIP_R_8F_E5M2;

  auto D_type = HIP_R_32F;
  auto bias_type = HIP_R_16BF;
  hipStream_t stream = 0;

  transformer_engine::cublas_gemm(A,
                                  A_scale_inv,
                                  B,
                                  B_scale_inv,
                                  D,
                                  /*bias_ptr=*/bias_ptr,
                                  /*pre_gelu_out=*/nullptr,
                                  m, n, k,
                                  lda, ldb, ldd,
                                  A_type,
                                  B_type,
                                  D_type,
                                  bias_type,
                                  (transa) ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                                  (transb) ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                                  /*bias=*/bias_ptr != nullptr,
                                  /*gelu=*/false,
                                  /*grad=*/false,
                                  workspace,
                                  workspaceSize,
                                  /*use_fp8=*/true,
                                  accumulate,
                                  use_split_accumulate,
                                  stream);

  checkCUDA(hipMemcpy(D_ptr, D, D_N * sizeof(DType),
                       hipMemcpyDeviceToHost));

  checkCUDA(hipFree(A));
  checkCUDA(hipFree(B));
  checkCUDA(hipFree(D));
  checkCUDA(hipFree(A_scale_inv));
  checkCUDA(hipFree(B_scale_inv));
  // checkCUDA(hipFree(bias_ptr));
  checkCUDA(hipFree(workspace));
  
}

}

